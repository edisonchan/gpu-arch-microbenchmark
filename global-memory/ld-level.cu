#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

using std::cout;
using std::endl;

__global__ void ld_global(){

}

int main() {
    int m = 256;
    size_t width = m * m;
    size_t bytes = 4 * width;

    dim3 blockDim(128);
    dim3 gridDim(2);

    float *A, *B, *C;
    float *h_A, *h_B, *h_C;

    hipMallocManaged(&A, bytes);
    hipMallocManaged(&B, bytes);
    hipMallocManaged(&C, bytes);

    h_A = malloc(bytes);
    h_B = malloc(bytes);
    h_C = malloc(bytes);
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // for (int i = 0; i < width; ++i) {
    //     A[i] = i;
    // }
    printf("\n");
    float       totalElapsed;
    hipEvent_t start_t, stop_t;
    hipEventCreate(&start_t);
    hipEventCreate(&stop_t);

    hipEventRecord(start_t, 0);

    sgemm_128x128_nt_cuda<<<gridDim, blockDim>>>(d_A, d_B, d_C, m, m, m);
    printf(hipGetErrorString(hipGetLastError()));

    hipEventRecord(stop_t, 0);
    hipEventSynchronize(stop_t);

    hipEventElapsedTime(&totalElapsed, start_t, stop_t);

    printf("\nTime Elapsed %f ms", totalElapsed);
    return 0;
}
