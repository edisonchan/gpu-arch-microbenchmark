
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <nvbit.h>

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char* event_name, void* params,
                         hipError_t* pStatus){
    printf(event_name);
}
