
#include <iostream>
#include <hip/hip_runtime.h>


void format_array(float *array, int size, int split_line){
    printf("\n");
    for (int i = 0; i < size; ++i){
        printf("%.2f, ", array[i]);
        if (i % split_line == split_line - 1){
            printf("\n");
        }
    }
    printf("\n");
}


void format_array(uint32_t *array, int size, int split_line){
    printf("\n");
    for (int i = 0; i < size; ++i){
        printf("%3u, ", array[i]);
        if (i % split_line == split_line - 1){
            printf("\n");
        }
    }
    printf("\n");
}


void format_array(int32_t *array, int size, int split_line){
    printf("\n");
    for (int i = 0; i < size; ++i){
        printf("%3d, ", array[i]);
        if (i % split_line == split_line - 1){
            printf("\n");
        }
    }
    printf("\n");
}


int main() {
    size_t width = 1024;
    size_t bytes = 4 * width;
    const char* cubin = "ld_global_cache.cubin";

    int32_t *h_A, *d_A;
    int32_t *h_cost, *d_cost;

    h_A = static_cast<int32_t*>(malloc(bytes));
    h_cost = static_cast<int32_t*>(malloc(bytes));
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_cost, bytes);

    for (int i = 0; i < width; ++i) {
        h_A[i] = i;
    }

    float       totalElapsed;
    hipEvent_t start_t, stop_t;

    hipModule_t module;
    hipFunction_t kernel;

    hipEventCreate(&start_t);
    hipEventCreate(&stop_t);

    hipModuleLoad(&module, cubin);
    hipModuleGetFunction(&kernel, module, "kern");

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipEventRecord(start_t, 0);

    void* args[2] = {&d_A, &d_cost};
    hipModuleLaunchKernel(kernel, 1, 1, 1,
                   1, 1, 1,
                   0, 0, args, 0);

    hipEventRecord(stop_t, 0);
    hipEventSynchronize(stop_t);
    printf(hipGetErrorString(hipGetLastError()));
    hipEventElapsedTime(&totalElapsed, start_t, stop_t);
    hipMemcpy(h_cost, d_cost, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_A, d_A, bytes, hipMemcpyDeviceToHost);

    printf("ld_global:");
    printf("Array A: \n");
    format_array(h_A, 32, 10);
    printf("Array Cost: \n");
    format_array(h_cost, 32, 10);
    printf("Host Time Elapsed %f ms\n", totalElapsed);
    printf("Throughput  %.3fGFlops", static_cast<float>(width) / totalElapsed / 1024 / 1024);
    return 0;
}
