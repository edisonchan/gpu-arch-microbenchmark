// 
//
//
//

#include "hip/hip_runtime.h"
#include "utils.cuh"


int main(){

    float* input_h; 
    float* input_d;
    float* output_h;
    float* output_d;
    uint32_t* clock_h;
    uint32_t* clock_d;

    int size = 1024;

    input_h     = static_cast<float*>(malloc(sizeof(float) * size));
    output_h    = static_cast<float*>(malloc(sizeof(float) * size));
    clock_h     = static_cast<uint32_t*>(malloc(sizeof(uint32_t) * size));


    hipMalloc(&input_d,  sizeof(float) * size);
    hipMalloc(&output_d, sizeof(float) * size);
    hipMalloc(&clock_d,  sizeof(uint32_t) * size);

    hipMemcpy(input_d, input_h, sizeof(float) * size, hipMemcpyHostToDevice);


    dim3 gDim(1, 1, 1);
    dim3 bDim(1, 1, 1);

    void* kernel_args[3] = {&input_d, &output_d, &clock_d};


    const char* cubin_name1 = "../sass_cubin/reg_with_bankconflict_75.cubin";
    const char* kernel_name1 = "regWithBankConflict";
    launchSassKernel(cubin_name1, kernel_name1, gDim, bDim, kernel_args);
    hipMemcpy(clock_h, clock_d, sizeof(uint) * size, hipMemcpyDeviceToHost);

    printf(">>> SASS-Level Reg With    BankConflict IPC Result\n");
    printf("        FFMA per \t%.3f cycle\n", static_cast<float>(clock_h[0]) / 128);
    



    const char* cubin_name2 = "../sass_cubin/reg_without_bankconflict_75.cubin";
    const char* kernel_name2 = "regWithoutBankConflict";
    launchSassKernel(cubin_name2, kernel_name2, gDim, bDim, kernel_args);
    hipMemcpy(clock_h, clock_d, sizeof(uint) * size, hipMemcpyDeviceToHost);

    printf(">>> SASS-Level Reg Without BankConflict IPC Result\n");
    printf("        FFMA per \t%.3f cycle\n", static_cast<float>(clock_h[0]) / 128);

    return 0;
}