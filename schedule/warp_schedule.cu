// warp exec position

#include <iostream>
#include <hip/hip_runtime.h>

__forceinline__ __device__ uint32_t get_warpid(){
    uint32_t clock;
    asm volatile(
        "mov.u32    %0,     %%warpid; \n\t"
        :"=r"(clock)::"memory"
    );
    return clock;
}
template <int M, int N>
__global__ void warp_workload(float *A, float *B){
    int tid = threadIdx.x;
    int warpid = get_warpid();
    if (warpid == M or warpid == N){
        float dummy = 0;
        float vA[4], vB[4], vC[4], vD[4];
        float *ptr;
        ptrdiff_t offset = 0;

        #pragma unroll
        for (int i = 0; i < 32; ++i){
            offset = i * 4;
            ptr = A + offset;

            asm volatile(
                "ld.global.ca.f32   %0,     [%4];       \n\t"
                "ld.global.ca.f32   %1,     [%4+4];     \n\t"
                "ld.global.ca.f32   %2,     [%4+8];     \n\t"
                "ld.global.ca.f32   %3,     [%4+12];    \n\t"
                :"=f"(vA[0]),"=f"(vB[0]),"=f"(vC[0]),"=f"(vD[0])
                :"l"(ptr):"memory"
            );
            dummy += vA[0];
            dummy += vB[0];
            dummy += vC[0];
            dummy += vD[0];
        }
        B[tid] = dummy;
    }
}


int main() {
    size_t width = 512;
    size_t bytes = 4 * width;

    dim3 bDim(256);
    dim3 gDim(1);

    float *h_A, *h_B;
    float *d_A, *d_B;
    uint32_t *h_cost, *d_cost;

    h_A = static_cast<float*>(malloc(bytes));
    h_B = static_cast<float*>(malloc(bytes));
    h_cost = static_cast<uint32_t*>(malloc(bytes));
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_cost, bytes);

    for (int i = 0; i < width; ++i) {
        h_A[i] = i;
    }

    float       totalElapsed;
    hipEvent_t start_t, stop_t;
    hipEventCreate(&start_t);
    hipEventCreate(&stop_t);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipEventRecord(start_t, 0);

    warp_workload<0, 1><<<gDim, bDim>>>(d_A, d_B);
    printf(hipGetErrorString(hipGetLastError()));

    hipEventRecord(stop_t, 0);
    hipEventSynchronize(stop_t);
    hipMemcpy(h_B, d_B, bytes, hipMemcpyDeviceToHost);
    hipEventElapsedTime(&totalElapsed, start_t, stop_t);
    printf("\nHost Time Elapsed %f ms", totalElapsed);


    hipEventRecord(start_t, 0);

    warp_workload<0, 2><<<gDim, bDim>>>(d_A, d_B);
    printf(hipGetErrorString(hipGetLastError()));

    hipEventRecord(stop_t, 0);
    hipEventSynchronize(stop_t);
    hipMemcpy(h_B, d_B, bytes, hipMemcpyDeviceToHost);

    hipEventElapsedTime(&totalElapsed, start_t, stop_t);
    printf("\nHost Time Elapsed %f ms", totalElapsed);





    hipEventRecord(start_t, 0);

    warp_workload<0, 3><<<gDim, bDim>>>(d_A, d_B);
    printf(hipGetErrorString(hipGetLastError()));

    hipEventRecord(stop_t, 0);
    hipEventSynchronize(stop_t);
    hipMemcpy(h_B, d_B, bytes, hipMemcpyDeviceToHost);

    hipEventElapsedTime(&totalElapsed, start_t, stop_t);
    printf("\nHost Time Elapsed %f ms", totalElapsed);




    hipEventRecord(start_t, 0);
    warp_workload<0, 4><<<gDim, bDim>>>(d_A, d_B);
    printf(hipGetErrorString(hipGetLastError()));

    hipEventRecord(stop_t, 0);
    hipEventSynchronize(stop_t);
    hipMemcpy(h_B, d_B, bytes, hipMemcpyDeviceToHost);

    hipEventElapsedTime(&totalElapsed, start_t, stop_t);
    printf("\nHost Time Elapsed %f ms", totalElapsed);





    hipEventRecord(start_t, 0);
    warp_workload<0, 5><<<gDim, bDim>>>(d_A, d_B);
    printf(hipGetErrorString(hipGetLastError()));

    hipEventRecord(stop_t, 0);
    hipEventSynchronize(stop_t);
    hipMemcpy(h_B, d_B, bytes, hipMemcpyDeviceToHost);

    hipEventElapsedTime(&totalElapsed, start_t, stop_t);
    printf("\nHost Time Elapsed %f ms", totalElapsed);






    hipEventRecord(start_t, 0);
    warp_workload<0, 6><<<gDim, bDim>>>(d_A, d_B);
    printf(hipGetErrorString(hipGetLastError()));

    hipEventRecord(stop_t, 0);
    hipEventSynchronize(stop_t);
    hipMemcpy(h_B, d_B, bytes, hipMemcpyDeviceToHost);

    hipEventElapsedTime(&totalElapsed, start_t, stop_t);
    printf("\nHost Time Elapsed %f ms", totalElapsed);





    hipEventRecord(start_t, 0);
    warp_workload<0, 7><<<gDim, bDim>>>(d_A, d_B);
    printf(hipGetErrorString(hipGetLastError()));

    hipEventRecord(stop_t, 0);
    hipEventSynchronize(stop_t);
    hipMemcpy(h_B, d_B, bytes, hipMemcpyDeviceToHost);

    hipEventElapsedTime(&totalElapsed, start_t, stop_t);
    printf("\nHost Time Elapsed %f ms", totalElapsed);





    return 0;
}
